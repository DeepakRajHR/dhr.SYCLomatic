// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(hipDeviceptr_t ptr, size_t size, size_t offset,
          unsigned long long flags) {
  // Start
  hipMemGenericAllocationHandle_t handle;
  hipMemMap(ptr /*CUdeviceptr*/, size /*size_t*/, offset /*size_t*/,
           handle /*CUmemGenericAllocationHandle*/,
           flags /*unsigned long long */);
  // End
}