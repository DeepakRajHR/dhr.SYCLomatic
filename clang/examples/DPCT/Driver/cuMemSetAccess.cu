// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(hipDeviceptr_t ptr, size_t size, hipMemAccessDesc *desc, size_t count) {
  // Start
  hipMemSetAccess(ptr /*CUdeviceptr*/, size /*size_t*/,
                 desc /*CUmemAccessDesc **/, count /*size_t*/);
  // End
}