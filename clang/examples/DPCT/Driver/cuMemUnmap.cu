// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(hipDeviceptr_t ptr, size_t size) {
  // Start
  hipMemUnmap(ptr /*CUdeviceptr*/, size /*size_t*/);
  // End
}