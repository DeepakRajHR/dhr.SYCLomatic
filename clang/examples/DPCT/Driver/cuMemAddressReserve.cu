// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(hipDeviceptr_t *ptr, size_t size, size_t alignment, hipDeviceptr_t addr,
          unsigned long long flags) {
  // Start
  hipMemAddressReserve(ptr /*CUdeviceptr **/, size /*size_t*/,
                      alignment /*size_t*/, addr /*CUdeviceptr*/,
                      flags /*unsigned long long*/);
  // End
}