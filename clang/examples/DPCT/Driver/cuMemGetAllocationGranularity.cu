// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(size_t *granularity, hipMemAllocationProp *prop,
          hipMemAllocationGranularity_flags option) {
  // Start
  hipMemGetAllocationGranularity(granularity /*size_t
                                             **/
                                ,
                                prop /*CUmemAllocationProp **/,
                                option /*CUmemAllocationGranularity_flags*/);
  // End
}