// Option: --use-experimental-features=virtual_mem
#include <hip/hip_runtime.h>
void test(size_t size, hipMemAllocationProp *prop, unsigned long long flags) {
  // Start
  hipMemGenericAllocationHandle_t *handle;
  hipMemCreate(handle /*CUmemGenericAllocationHandle **/, size /*size_t*/,
              prop /*CUmemAllocationProp **/, flags /*unsigned long long*/);
  // End
}