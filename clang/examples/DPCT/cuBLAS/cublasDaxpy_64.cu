
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const double *alpha,
          const double *x, int64_t incx, double *y, int64_t incy) {
  // Start
  hipblasDaxpy_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const double **/, x /*const double **/,
                 incx /*int64_t*/, y /*double **/, incy /*int64_t*/);
  // End
}
