#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -format-range=none -in-root %S -out-root %T/blocklevel/blockshuffle %S/blockshuffle.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/blocklevel/blockshuffle/blockshuffle.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/blocklevel/blockshuffle/blockshuffle.dp.cpp -o %T/blocklevel/blockshuffle/blockshuffle.dp.o %}

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cub/block/block_shuffle.cuh>

// CHECK: void BlockShuffleKernel(const sycl::nd_item<3> &item_ct1,
// CHECK:       uint8_t *temp_storage) {
// CHECK:   int d[4];
// CHECK:   typedef dpct::group::group_shuffle<int, 128, 1> BS;
// CHECK:   int a;
// CHECK:   BS(temp_storage).select(item_ct1, a, a, 2);
// CHECK:   BS(temp_storage).select2(item_ct1, a, a, 2);
// CHECK:   BS(temp_storage).shuffle_right(item_ct1, d, d, a);
// CHECK:   BS(temp_storage).shuffle_left(item_ct1, d, d);
// CHECK: }

__global__ void BlockShuffleKernel() {
  int d[4];
  typedef hipcub::BlockShuffle<int, 128, 1> BS;
  __shared__ typename BS::TempStorage temp_storage;
  int a;
  BS(temp_storage).Offset(a, a, 2);
  BS(temp_storage).Rotate(a, a, 2);
  BS(temp_storage).Up<4>(d, d, a);
  BS(temp_storage).Down<4>(d, d);
}

bool test_striped_to_blocked() {
// CHECK: dpct::get_in_order_queue().submit(
// CHECK:       [&](sycl::handler &cgh) {
// CHECK:         sycl::local_accessor<uint8_t, 1> temp_storage_acc(dpct::group::group_shuffle<int, 128>::get_local_memory_size(sycl::range<3>(1, 1, 128).size()), cgh);
// CHECK:         cgh.parallel_for(
// CHECK:           sycl::nd_range<3>(sycl::range<3>(1, 1, 128), sycl::range<3>(1, 1, 128)),
// CHECK:           [=](sycl::nd_item<3> item_ct1) {
// CHECK:             BlockShuffleKernel(item_ct1, &temp_storage_acc[0]);
// CHECK:           });
// CHECK:       });
  BlockShuffleKernel<<<1, 128>>>();
  hipDeviceSynchronize();

  return true;
}

int main() {
  test_striped_to_blocked();
  return 0;
};
