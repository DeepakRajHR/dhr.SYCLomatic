#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/template_uninstantiated %S/template_uninstantiated.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/template_uninstantiated/template_uninstantiated.dp.cpp --match-full-lines %s

#include <hipcub/hipcub.hpp>


// CHECK: template <typename T>
// CHECK: void kernel() {
// CHECK:   typedef sycl::group<3> BS;
// CHECK-NOT: typename BS::TempStorage temp_storage;
// CHECK:   T thread_data, output;
// CHECK:   /*
// CHECK:   DPCT1028:{{[0-9]+}}: The ExclusiveSum member function call was not migrated because the caller function may not instantiated.
// CHECK:   */
// CHECK:   BS(temp_storage).ExclusiveSum(thread_data, output);
// CHECK: }
template <typename T>
__global__ void kernel() {

  typedef hipcub::BlockScan<T, 128> BS;

  __shared__ typename BS::TempStorage temp_storage;

  T thread_data, output;

  BS(temp_storage).ExclusiveSum(thread_data, output);
}
