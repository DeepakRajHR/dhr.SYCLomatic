// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --format-range=none -out-root %T/cuGraphics_default_option %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuGraphics_default_option/cuGraphics_default_option.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -DNO_BUILD_TEST -fsycl %T/cuGraphics_default_option/cuGraphics_default_option.dp.cpp -o %T/cuGraphics_default_option/cudaGraphicsResource_test.dp.o %}

#ifndef NO_BUILD_TEST
#include <hip/hip_runtime.h>

int main() {
  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of CUgraphicsResource is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsResource_t resource, *resources;

  hipDeviceptr_t devPtr;
  size_t size;

  hipStream_t stream;

  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of cuGraphicsMapResources is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsMapResources(2, resources, stream);

  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of cuGraphicsMapResources is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsMapResources(1, &resource, stream);

  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of cuGraphicsResourceGetMappedPointer_v2 is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsResourceGetMappedPointer(&devPtr, &size, resource);

  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of cuGraphicsUnmapResources is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsUnmapResources(2, resources, stream);

  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of cuGraphicsUnmapResources is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsUnmapResources(1, &resource, stream);

  // CHECK: /*
  // CHECK-NEXT: DPCT1119:{{[0-9]+}}: Migration of cuGraphicsUnregisterResource is not supported, please try to remigrate with option: --use-experimental-features=bindless_images.
  // CHECK-NEXT: */
  hipGraphicsUnregisterResource(resource);

  return 0;
}

#endif
