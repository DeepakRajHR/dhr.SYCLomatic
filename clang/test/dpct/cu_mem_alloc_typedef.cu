// UNSUPPORTED: system-linux
// RUN: dpct --format-range=none -out-root %T/cu_mem_alloc_typedef %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cu_mem_alloc_typedef/cu_mem_alloc_typedef.dp.cpp
#include <cstdint>
#include <hip/hip_runtime.h>

typedef uint64_t hipDeviceptr_t;

void foo(hipDeviceptr_t ptr) {
  ptr = hipMalloc(&ptr, 1024);
}
