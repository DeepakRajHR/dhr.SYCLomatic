// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct -out-root %T/driver-stream-and-event_after8.0 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-stream-and-event_after8.0/driver-stream-and-event_after8.0.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/driver-stream-and-event_after8.0/driver-stream-and-event_after8.0.dp.cpp -o %T/driver-stream-and-event_after8.0/driver-stream-and-event_after8.0.dp.o %}

#include "hip/hip_runtime.h"

// CHECK: void test_stream_and_context(dpct::queue_ptr stream, int &context) {
void test_stream_and_context(hipStream_t stream, hipCtx_t& context) {
  // CHECK: context = dpct::get_device_id(stream->get_device());
  cuStreamGetCtx(stream, &context);
}
