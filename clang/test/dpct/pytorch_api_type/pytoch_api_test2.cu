// RUN: dpct --rule-file=%S/../../../tools/dpct/DpctOptRules/pytorch_api.yaml --format-range=none -out-root %T/pytoch_api_test2 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/pytoch_api_test2/pytoch_api_test2.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define AT_CUDA_CHECK(stmt)  (stmt)

namespace c10 {
using DeviceIndex = int8_t;
namespace cuda {
class CUDAStream {
public:
  CUDAStream() {}
  hipStream_t stream() { return 0; }
  operator hipStream_t() const {
    return stream();
  }
  hipStream_t stream() const;
};
CUDAStream getCurrentCUDAStream(DeviceIndex device_index = -1) {
  return CUDAStream();
}
} // namespace cuda
} // namespace c10

namespace at {
using namespace c10;
} // namespace at

__global__ void kernel() {}

int main() {
  dim3 gridSize(2, 2, 1);
  dim3 blockSize(8, 8, 1);
  void *args[] = {nullptr}; 

  //      CHECK:([&](){
  // CHECK-NEXT:  ((sycl::queue*)(c10::xpu::getCurrentXPUStream()))->parallel_for(
  // CHECK-NEXT:    sycl::nd_range<3>(gridSize * blockSize, blockSize),
  // CHECK-NEXT:    [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:      kernel();
  // CHECK-NEXT:    });
  // CHECK-NEXT:  return 0;
  // CHECK-NEXT:}());
  AT_CUDA_CHECK(hipLaunchKernel(reinterpret_cast<const void*>((const void *)kernel), gridSize, blockSize, args, 0, at::cuda::getCurrentCUDAStream()));
  at::DeviceIndex d = 1;
  // CHECK: c10::xpu::getCurrentXPUStream(d);
  at::cuda::getCurrentCUDAStream(d);
  // CHECK: dpct::queue_ptr s = &static_cast<sycl::queue &>(c10::xpu::getCurrentXPUStream(). queue());
  hipStream_t s = at::cuda::getCurrentCUDAStream().stream();
  return 0;
}
