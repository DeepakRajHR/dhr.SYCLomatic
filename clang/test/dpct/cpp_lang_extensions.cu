#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/cpp_lang_extensions %s --cuda-include-path="%cuda-path/include" -extra-arg="-I%S" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cpp_lang_extensions/cpp_lang_extensions.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cpp_lang_extensions/cpp_lang_extensions.dp.cpp -o %T/cpp_lang_extensions/cpp_lang_extensions.dp.o %}

#include "cpp_lang_extensions.cuh"

__device__ float df(float f) {
  float a[23];
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f;
  __ldg(&f);
  int *pi;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *pi;
  __ldg(pi);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(pi + 2);
  __ldg(pi + 2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: return 45 * a[23] * f * 23;
  return 45 * __ldg(&a[23]) * f * 23;
}

// CHECK: /*
// CHECK-NEXT: DPCT1110:{{[0-9]+}}: The total declared local variable size in device function dev exceeds 128 bytes and may cause high register pressure. Consult with your hardware vendor to find the total register size available and adjust the code, or use smaller sub-group size to avoid high register pressure.
// CHECK-NEXT: */
__device__ void dev() {
  char *c_1;
  char c_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c_2 = *c_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c_2 = *c_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c_2 = *c_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c_2 = *c_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c_2 = *c_1;
  c_2 = __ldcg(c_1);
  c_2 = __ldca(c_1);
  c_2 = __ldcs(c_1);
  c_2 = __ldlu(c_1);
  c_2 = __ldcv(c_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c_1 = c_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c_1 = c_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c_1 = c_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c_1 = c_2;
  __stwb(c_1, c_2);
  __stcg(c_1, c_2);
  __stcs(c_1, c_2);
  __stwt(c_1, c_2);

  char2 *c2_1;
  char2 c2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c2_2 = *c2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c2_2 = *c2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c2_2 = *c2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c2_2 = *c2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c2_2 = *c2_1;
  c2_2 = __ldcg(c2_1);
  c2_2 = __ldca(c2_1);
  c2_2 = __ldcs(c2_1);
  c2_2 = __ldlu(c2_1);
  c2_2 = __ldcv(c2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c2_1 = c2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c2_1 = c2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c2_1 = c2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c2_1 = c2_2;
  __stwb(c2_1, c2_2);
  __stcg(c2_1, c2_2);
  __stcs(c2_1, c2_2);
  __stwt(c2_1, c2_2);

  char4 *c4_1;
  char4 c4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c4_2 = *c4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c4_2 = *c4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c4_2 = *c4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c4_2 = *c4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: c4_2 = *c4_1;
  c4_2 = __ldcg(c4_1);
  c4_2 = __ldca(c4_1);
  c4_2 = __ldcs(c4_1);
  c4_2 = __ldlu(c4_1);
  c4_2 = __ldcv(c4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c4_1 = c4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c4_1 = c4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c4_1 = c4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *c4_1 = c4_2;
  __stwb(c4_1, c4_2);
  __stcg(c4_1, c4_2);
  __stcs(c4_1, c4_2);
  __stwt(c4_1, c4_2);

  double *d_1;
  double d_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d_2 = *d_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d_2 = *d_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d_2 = *d_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d_2 = *d_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d_2 = *d_1;
  d_2 = __ldcg(d_1);
  d_2 = __ldca(d_1);
  d_2 = __ldcs(d_1);
  d_2 = __ldlu(d_1);
  d_2 = __ldcv(d_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d_1 = d_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d_1 = d_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d_1 = d_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d_1 = d_2;
  __stwb(d_1, d_2);
  __stcg(d_1, d_2);
  __stcs(d_1, d_2);
  __stwt(d_1, d_2);

  double2 *d2_1;
  double2 d2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2_2 = *d2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2_2 = *d2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2_2 = *d2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2_2 = *d2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2_2 = *d2_1;
  d2_2 = __ldcg(d2_1);
  d2_2 = __ldca(d2_1);
  d2_2 = __ldcs(d2_1);
  d2_2 = __ldlu(d2_1);
  d2_2 = __ldcv(d2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d2_1 = d2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d2_1 = d2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d2_1 = d2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *d2_1 = d2_2;
  __stwb(d2_1, d2_2);
  __stcg(d2_1, d2_2);
  __stcs(d2_1, d2_2);
  __stwt(d2_1, d2_2);

  float *f_1;
  float f_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f_2 = *f_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f_2 = *f_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f_2 = *f_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f_2 = *f_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f_2 = *f_1;
  f_2 = __ldcg(f_1);
  f_2 = __ldca(f_1);
  f_2 = __ldcs(f_1);
  f_2 = __ldlu(f_1);
  f_2 = __ldcv(f_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f_1 = f_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f_1 = f_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f_1 = f_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f_1 = f_2;
  __stwb(f_1, f_2);
  __stcg(f_1, f_2);
  __stcs(f_1, f_2);
  __stwt(f_1, f_2);

  float2 *f2_1;
  float2 f2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2_2 = *f2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2_2 = *f2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2_2 = *f2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2_2 = *f2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2_2 = *f2_1;
  f2_2 = __ldcg(f2_1);
  f2_2 = __ldca(f2_1);
  f2_2 = __ldcs(f2_1);
  f2_2 = __ldlu(f2_1);
  f2_2 = __ldcv(f2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f2_1 = f2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f2_1 = f2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f2_1 = f2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f2_1 = f2_2;
  __stwb(f2_1, f2_2);
  __stcg(f2_1, f2_2);
  __stcs(f2_1, f2_2);
  __stwt(f2_1, f2_2);

  float4 *f4_1;
  float4 f4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f4_2 = *f4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f4_2 = *f4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f4_2 = *f4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f4_2 = *f4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: f4_2 = *f4_1;
  f4_2 = __ldcg(f4_1);
  f4_2 = __ldca(f4_1);
  f4_2 = __ldcs(f4_1);
  f4_2 = __ldlu(f4_1);
  f4_2 = __ldcv(f4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f4_1 = f4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f4_1 = f4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f4_1 = f4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *f4_1 = f4_2;
  __stwb(f4_1, f4_2);
  __stcg(f4_1, f4_2);
  __stcs(f4_1, f4_2);
  __stwt(f4_1, f4_2);

  int *i_1;
  int i_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i_2 = *i_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i_2 = *i_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i_2 = *i_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i_2 = *i_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i_2 = *i_1;
  i_2 = __ldcg(i_1);
  i_2 = __ldca(i_1);
  i_2 = __ldcs(i_1);
  i_2 = __ldlu(i_1);
  i_2 = __ldcv(i_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i_1 = i_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i_1 = i_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i_1 = i_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i_1 = i_2;
  __stwb(i_1, i_2);
  __stcg(i_1, i_2);
  __stcs(i_1, i_2);
  __stwt(i_1, i_2);

  int2 *i2_1;
  int2 i2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i2_2 = *i2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i2_2 = *i2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i2_2 = *i2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i2_2 = *i2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i2_2 = *i2_1;
  i2_2 = __ldcg(i2_1);
  i2_2 = __ldca(i2_1);
  i2_2 = __ldcs(i2_1);
  i2_2 = __ldlu(i2_1);
  i2_2 = __ldcv(i2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i2_1 = i2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i2_1 = i2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i2_1 = i2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i2_1 = i2_2;
  __stwb(i2_1, i2_2);
  __stcg(i2_1, i2_2);
  __stcs(i2_1, i2_2);
  __stwt(i2_1, i2_2);

  int4 *i4_1;
  int4 i4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i4_2 = *i4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i4_2 = *i4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i4_2 = *i4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i4_2 = *i4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: i4_2 = *i4_1;
  i4_2 = __ldcg(i4_1);
  i4_2 = __ldca(i4_1);
  i4_2 = __ldcs(i4_1);
  i4_2 = __ldlu(i4_1);
  i4_2 = __ldcv(i4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i4_1 = i4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i4_1 = i4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i4_1 = i4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *i4_1 = i4_2;
  __stwb(i4_1, i4_2);
  __stcg(i4_1, i4_2);
  __stcs(i4_1, i4_2);
  __stwt(i4_1, i4_2);

  long *l_1;
  long l_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: l_2 = *l_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: l_2 = *l_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: l_2 = *l_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: l_2 = *l_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: l_2 = *l_1;
  l_2 = __ldcg(l_1);
  l_2 = __ldca(l_1);
  l_2 = __ldcs(l_1);
  l_2 = __ldlu(l_1);
  l_2 = __ldcv(l_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *l_1 = l_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *l_1 = l_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *l_1 = l_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *l_1 = l_2;
  __stwb(l_1, l_2);
  __stcg(l_1, l_2);
  __stcs(l_1, l_2);
  __stwt(l_1, l_2);

  long long *ll_1;
  long long ll_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll_2 = *ll_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll_2 = *ll_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll_2 = *ll_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll_2 = *ll_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll_2 = *ll_1;
  ll_2 = __ldcg(ll_1);
  ll_2 = __ldca(ll_1);
  ll_2 = __ldcs(ll_1);
  ll_2 = __ldlu(ll_1);
  ll_2 = __ldcv(ll_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll_1 = ll_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll_1 = ll_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll_1 = ll_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll_1 = ll_2;
  __stwb(ll_1, ll_2);
  __stcg(ll_1, ll_2);
  __stcs(ll_1, ll_2);
  __stwt(ll_1, ll_2);

  longlong2 *ll2_1;
  longlong2 ll2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll2_2 = *ll2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll2_2 = *ll2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll2_2 = *ll2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll2_2 = *ll2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ll2_2 = *ll2_1;
  ll2_2 = __ldcg(ll2_1);
  ll2_2 = __ldca(ll2_1);
  ll2_2 = __ldcs(ll2_1);
  ll2_2 = __ldlu(ll2_1);
  ll2_2 = __ldcv(ll2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll2_1 = ll2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll2_1 = ll2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll2_1 = ll2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ll2_1 = ll2_2;
  __stwb(ll2_1, ll2_2);
  __stcg(ll2_1, ll2_2);
  __stcs(ll2_1, ll2_2);
  __stwt(ll2_1, ll2_2);

  short *s_1;
  short s_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s_2 = *s_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s_2 = *s_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s_2 = *s_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s_2 = *s_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s_2 = *s_1;
  s_2 = __ldcg(s_1);
  s_2 = __ldca(s_1);
  s_2 = __ldcs(s_1);
  s_2 = __ldlu(s_1);
  s_2 = __ldcv(s_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s_1 = s_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s_1 = s_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s_1 = s_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s_1 = s_2;
  __stwb(s_1, s_2);
  __stcg(s_1, s_2);
  __stcs(s_1, s_2);
  __stwt(s_1, s_2);

  short2 *s2_1;
  short2 s2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2_2 = *s2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2_2 = *s2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2_2 = *s2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2_2 = *s2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2_2 = *s2_1;
  s2_2 = __ldcg(s2_1);
  s2_2 = __ldca(s2_1);
  s2_2 = __ldcs(s2_1);
  s2_2 = __ldlu(s2_1);
  s2_2 = __ldcv(s2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s2_1 = s2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s2_1 = s2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s2_1 = s2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s2_1 = s2_2;
  __stwb(s2_1, s2_2);
  __stcg(s2_1, s2_2);
  __stcs(s2_1, s2_2);
  __stwt(s2_1, s2_2);

  short4 *s4_1;
  short4 s4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s4_2 = *s4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s4_2 = *s4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s4_2 = *s4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s4_2 = *s4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: s4_2 = *s4_1;
  s4_2 = __ldcg(s4_1);
  s4_2 = __ldca(s4_1);
  s4_2 = __ldcs(s4_1);
  s4_2 = __ldlu(s4_1);
  s4_2 = __ldcv(s4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s4_1 = s4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s4_1 = s4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s4_1 = s4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *s4_1 = s4_2;
  __stwb(s4_1, s4_2);
  __stcg(s4_1, s4_2);
  __stcs(s4_1, s4_2);
  __stwt(s4_1, s4_2);

  signed char *sc_1;
  signed char sc_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: sc_2 = *sc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: sc_2 = *sc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: sc_2 = *sc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: sc_2 = *sc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: sc_2 = *sc_1;
  sc_2 = __ldcg(sc_1);
  sc_2 = __ldca(sc_1);
  sc_2 = __ldcs(sc_1);
  sc_2 = __ldlu(sc_1);
  sc_2 = __ldcv(sc_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *sc_1 = sc_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *sc_1 = sc_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *sc_1 = sc_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *sc_1 = sc_2;
  __stwb(sc_1, sc_2);
  __stcg(sc_1, sc_2);
  __stcs(sc_1, sc_2);
  __stwt(sc_1, sc_2);

  uchar2 *uc2_1;
  uchar2 uc2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc2_2 = *uc2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc2_2 = *uc2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc2_2 = *uc2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc2_2 = *uc2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc2_2 = *uc2_1;
  uc2_2 = __ldcg(uc2_1);
  uc2_2 = __ldca(uc2_1);
  uc2_2 = __ldcs(uc2_1);
  uc2_2 = __ldlu(uc2_1);
  uc2_2 = __ldcv(uc2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc2_1 = uc2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc2_1 = uc2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc2_1 = uc2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc2_1 = uc2_2;
  __stwb(uc2_1, uc2_2);
  __stcg(uc2_1, uc2_2);
  __stcs(uc2_1, uc2_2);
  __stwt(uc2_1, uc2_2);

  uchar4 *uc4_1;
  uchar4 uc4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc4_2 = *uc4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc4_2 = *uc4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc4_2 = *uc4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc4_2 = *uc4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc4_2 = *uc4_1;
  uc4_2 = __ldcg(uc4_1);
  uc4_2 = __ldca(uc4_1);
  uc4_2 = __ldcs(uc4_1);
  uc4_2 = __ldlu(uc4_1);
  uc4_2 = __ldcv(uc4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc4_1 = uc4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc4_1 = uc4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc4_1 = uc4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc4_1 = uc4_2;
  __stwb(uc4_1, uc4_2);
  __stcg(uc4_1, uc4_2);
  __stcs(uc4_1, uc4_2);
  __stwt(uc4_1, uc4_2);

  uint2 *ui2_1;
  uint2 ui2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui2_2 = *ui2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui2_2 = *ui2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui2_2 = *ui2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui2_2 = *ui2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui2_2 = *ui2_1;
  ui2_2 = __ldcg(ui2_1);
  ui2_2 = __ldca(ui2_1);
  ui2_2 = __ldcs(ui2_1);
  ui2_2 = __ldlu(ui2_1);
  ui2_2 = __ldcv(ui2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui2_1 = ui2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui2_1 = ui2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui2_1 = ui2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui2_1 = ui2_2;
  __stwb(ui2_1, ui2_2);
  __stcg(ui2_1, ui2_2);
  __stcs(ui2_1, ui2_2);
  __stwt(ui2_1, ui2_2);

  uint4 *ui4_1;
  uint4 ui4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui4_2 = *ui4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui4_2 = *ui4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui4_2 = *ui4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui4_2 = *ui4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui4_2 = *ui4_1;
  ui4_2 = __ldcg(ui4_1);
  ui4_2 = __ldca(ui4_1);
  ui4_2 = __ldcs(ui4_1);
  ui4_2 = __ldlu(ui4_1);
  ui4_2 = __ldcv(ui4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui4_1 = ui4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui4_1 = ui4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui4_1 = ui4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui4_1 = ui4_2;
  __stwb(ui4_1, ui4_2);
  __stcg(ui4_1, ui4_2);
  __stcs(ui4_1, ui4_2);
  __stwt(ui4_1, ui4_2);

  ulonglong2 *ull2_1;
  ulonglong2 ull2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull2_2 = *ull2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull2_2 = *ull2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull2_2 = *ull2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull2_2 = *ull2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull2_2 = *ull2_1;
  ull2_2 = __ldcg(ull2_1);
  ull2_2 = __ldca(ull2_1);
  ull2_2 = __ldcs(ull2_1);
  ull2_2 = __ldlu(ull2_1);
  ull2_2 = __ldcv(ull2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull2_1 = ull2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull2_1 = ull2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull2_1 = ull2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull2_1 = ull2_2;
  __stwb(ull2_1, ull2_2);
  __stcg(ull2_1, ull2_2);
  __stcs(ull2_1, ull2_2);
  __stwt(ull2_1, ull2_2);

  unsigned char *uc_1;
  unsigned char uc_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc_2 = *uc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc_2 = *uc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc_2 = *uc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc_2 = *uc_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: uc_2 = *uc_1;
  uc_2 = __ldcg(uc_1);
  uc_2 = __ldca(uc_1);
  uc_2 = __ldcs(uc_1);
  uc_2 = __ldlu(uc_1);
  uc_2 = __ldcv(uc_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc_1 = uc_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc_1 = uc_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc_1 = uc_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *uc_1 = uc_2;
  __stwb(uc_1, uc_2);
  __stcg(uc_1, uc_2);
  __stcs(uc_1, uc_2);
  __stwt(uc_1, uc_2);

  unsigned int *ui_1;
  unsigned int ui_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui_2 = *ui_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui_2 = *ui_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui_2 = *ui_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui_2 = *ui_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ui_2 = *ui_1;
  ui_2 = __ldcg(ui_1);
  ui_2 = __ldca(ui_1);
  ui_2 = __ldcs(ui_1);
  ui_2 = __ldlu(ui_1);
  ui_2 = __ldcv(ui_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui_1 = ui_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui_1 = ui_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui_1 = ui_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ui_1 = ui_2;
  __stwb(ui_1, ui_2);
  __stcg(ui_1, ui_2);
  __stcs(ui_1, ui_2);
  __stwt(ui_1, ui_2);

  unsigned long *ul_1;
  unsigned long ul_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ul_2 = *ul_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ul_2 = *ul_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ul_2 = *ul_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ul_2 = *ul_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ul_2 = *ul_1;
  ul_2 = __ldcg(ul_1);
  ul_2 = __ldca(ul_1);
  ul_2 = __ldcs(ul_1);
  ul_2 = __ldlu(ul_1);
  ul_2 = __ldcv(ul_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ul_1 = ul_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ul_1 = ul_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ul_1 = ul_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ul_1 = ul_2;
  __stwb(ul_1, ul_2);
  __stcg(ul_1, ul_2);
  __stcs(ul_1, ul_2);
  __stwt(ul_1, ul_2);

  unsigned long long *ull_1;
  unsigned long long ull_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull_2 = *ull_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull_2 = *ull_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull_2 = *ull_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull_2 = *ull_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: ull_2 = *ull_1;
  ull_2 = __ldcg(ull_1);
  ull_2 = __ldca(ull_1);
  ull_2 = __ldcs(ull_1);
  ull_2 = __ldlu(ull_1);
  ull_2 = __ldcv(ull_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull_1 = ull_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull_1 = ull_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull_1 = ull_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *ull_1 = ull_2;
  __stwb(ull_1, ull_2);
  __stcg(ull_1, ull_2);
  __stcs(ull_1, ull_2);
  __stwt(ull_1, ull_2);

  unsigned short *us_1;
  unsigned short us_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us_2 = *us_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us_2 = *us_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us_2 = *us_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us_2 = *us_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us_2 = *us_1;
  us_2 = __ldcg(us_1);
  us_2 = __ldca(us_1);
  us_2 = __ldcs(us_1);
  us_2 = __ldlu(us_1);
  us_2 = __ldcv(us_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us_1 = us_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us_1 = us_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us_1 = us_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us_1 = us_2;
  __stwb(us_1, us_2);
  __stcg(us_1, us_2);
  __stcs(us_1, us_2);
  __stwt(us_1, us_2);

  ushort2 *us2_1;
  ushort2 us2_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us2_2 = *us2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us2_2 = *us2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us2_2 = *us2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us2_2 = *us2_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us2_2 = *us2_1;
  us2_2 = __ldcg(us2_1);
  us2_2 = __ldca(us2_1);
  us2_2 = __ldcs(us2_1);
  us2_2 = __ldlu(us2_1);
  us2_2 = __ldcv(us2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us2_1 = us2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us2_1 = us2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us2_1 = us2_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us2_1 = us2_2;
  __stwb(us2_1, us2_2);
  __stcg(us2_1, us2_2);
  __stcs(us2_1, us2_2);
  __stwt(us2_1, us2_2);

  ushort4 *us4_1;
  ushort4 us4_2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us4_2 = *us4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us4_2 = *us4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us4_2 = *us4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us4_2 = *us4_1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: us4_2 = *us4_1;
  us4_2 = __ldcg(us4_1);
  us4_2 = __ldca(us4_1);
  us4_2 = __ldcs(us4_1);
  us4_2 = __ldlu(us4_1);
  us4_2 = __ldcv(us4_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us4_1 = us4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us4_1 = us4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us4_1 = us4_2;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *us4_1 = us4_2;
  __stwb(us4_1, us4_2);
  __stcg(us4_1, us4_2);
  __stcs(us4_1, us4_2);
  __stwt(us4_1, us4_2);
}

#define DEV_INLINE __device__ __forceinline__
__device__ __constant__ uint2 const keccak_round_constants[4] = {
    { 0x80008081, 0x80000000 }, { 0x00008080, 0x80000000 }, { 0x80000001, 0x00000000 }, { 0x80008008, 0x80000000 }
};


// CHECK:namespace dpct_operator_overloading {
// CHECK:inline sycl::uint2 &operator^=(sycl::uint2 &v, const sycl::uint2 &v2) {
// CHECK:  return v;
// CHECK:}
// CHECK:}  // namespace dpct_operator_overloading
__host__ __device__ inline uint2 &operator^=(uint2 &v, const uint2 &v2) {
  return v;
}


DEV_INLINE void SHA3_512(uint2* s) {
    int i;
  // CHECK:    dpct_operator_overloading::operator^=(s[0] , LDG(keccak_round_constants[i]));
  // CHECK-NEXT:    LDG(keccak_round_constants[23]);
    s[0] ^= LDG(keccak_round_constants[i]);
    LDG(keccak_round_constants[23]);
}
