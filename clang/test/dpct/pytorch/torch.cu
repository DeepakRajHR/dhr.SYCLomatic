// RUN: rm -rf %T/pytorch/torch
// RUN: mkdir -p %T/pytorch/torch/src
// RUN: cp %S/torch.cu %T/pytorch/torch/src/
// RUN: cp -r %S/pytorch_inc %T/pytorch/torch/
// RUN: cd %T/pytorch/torch
// RUN: mkdir dpct_out
// RUN: dpct --out-root dpct_out %T/pytorch/torch/src/torch.cu --extra-arg="-I%T/pytorch/torch/pytorch_inc" --cuda-include-path="%cuda-path/include" --rule-file=%S/../../../tools/dpct/extensions/pytorch_api_rules/pytorch_api.yaml --analysis-scope-path %T/pytorch/torch/pytorch_inc --analysis-scope-path %T/pytorch/torch/src --in-root %T/pytorch/torch/src
// RUN: FileCheck --input-file %T/pytorch/torch/dpct_out/torch.dp.cpp --match-full-lines %T/pytorch/torch/src/torch.cu

// CHECK: #include <c10/xpu/XPUStream.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <torch/torch.h>

#define MY_CHECK(condition, message)                              \
  do {                                                            \
    if (!(condition)) {                                           \
      throw std::runtime_error("Error: " + std::string(message)); \
    }                                                             \
  } while (0)

// void foo(torch::Tensor x) {
void foo(torch::Tensor x) {
  // CHECK: MY_CHECK(x.is_xpu(), "x must reside on device");
  MY_CHECK(x.is_cuda(), "x must reside on device");
}
