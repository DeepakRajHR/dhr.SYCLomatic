// RUN: rm -rf %T/pytorch/c10
// RUN: mkdir -p %T/pytorch/c10/src
// RUN: cp %S/c10.cu %T/pytorch/c10/src/
// RUN: cp -r %S/pytorch_inc %T/pytorch/c10/
// RUN: cd %T/pytorch/c10
// RUN: mkdir dpct_out
// RUN: dpct -out-root dpct_out %T/pytorch/c10/src/c10.cu --extra-arg="-I%T/pytorch/c10/pytorch_inc" --cuda-include-path="%cuda-path/include" --rule-file=%S/../../../tools/dpct/DpctOptRules/pytorch_api.yaml  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/pytorch/c10/dpct_out/c10.dp.cpp --match-full-lines %T/pytorch/c10/src/c10.cu

#include <iostream>
// CHECK: #include <c10/core/DeviceGuard.h>
#include <c10/cuda/CUDAGuard.h>
// CHECK: #include <c10/xpu/XPUStream.h>
#include <c10/cuda/CUDAStream.h>
// CHECK: #include <c10/xpu/XPUFunctions.h>
#include <c10/cuda/CUDAFunctions.h>
// CHECK: #include <c10/xpu/XPUMacros.h>
#include <c10/cuda/CUDAMacros.h>

// CHECK: #define BE_BUILD_SHARED_LIBS C10_XPU_BUILD_SHARED_LIBS
// CHECK-NEXT: #define BE_EXPORT C10_XPU_EXPORT
// CHECK-NEXT: #define BE_IMPORT C10_XPU_IMPORT
// CHECK-NEXT: #define BE_API C10_XPU_API
// CHECK-NEXT: #define BE_BUILD_MAIN_LIB C10_XPU_BUILD_MAIN_LIB
#define BE_BUILD_SHARED_LIBS C10_CUDA_BUILD_SHARED_LIBS
#define BE_EXPORT C10_CUDA_EXPORT
#define BE_IMPORT C10_CUDA_IMPORT
#define BE_API C10_CUDA_API
#define BE_BUILD_MAIN_LIB C10_CUDA_BUILD_MAIN_LIB

int main() {
  // device APIs
  // CHECK: c10::DeviceIndex num_devices = c10::xpu::device_count();
  c10::DeviceIndex num_devices = c10::cuda::device_count();

  // CHECK: c10::DeviceIndex num_devices_ensured =
  // CHECK-NEXT:     c10::xpu::device_count_ensure_non_zero();
  c10::DeviceIndex num_devices_ensured = c10::cuda::device_count_ensure_non_zero();

  // CHECK: c10::DeviceIndex current_device = c10::xpu::current_device();
  c10::DeviceIndex current_device = c10::cuda::current_device();

  c10::DeviceIndex new_device = 1;
  // CHECK: c10::xpu::set_device(new_device);
  c10::cuda::set_device(new_device);

  // CHECK: c10::DeviceIndex exchanged_device = c10::xpu::exchange_device(0);
  c10::DeviceIndex exchanged_device = c10::cuda::ExchangeDevice(0);

  // CHECK: c10::DeviceIndex maybe_exchanged_device = c10::xpu::maybe_exchange_device(1);
  c10::DeviceIndex maybe_exchanged_device = c10::cuda::MaybeExchangeDevice(1);

  std::optional<c10::Device> device;
  try {
    // CHECK: c10::OptionalDeviceGuard device_guard(device);
    c10::cuda::OptionalCUDAGuard device_guard(device);
  } catch (const std::exception &e) {
    std::cerr << "Runtime error: " << e.what() << std::endl;
    return -1;
  }

  // stream APIs
  // CHECK: auto currentStream = c10::xpu::getCurrentXPUStream();
  auto currentStream = c10::cuda::getCurrentCUDAStream();

  // CHECK: dpct::queue_ptr curr_cuda_st =
  // CHECK-NEXT: &static_cast<sycl::queue &>(currentStream.queue());
  hipStream_t curr_cuda_st = currentStream.stream();
  // CHECK: curr_cuda_st =
  // CHECK-NEXT: &static_cast<sycl::queue &>(c10::xpu::getCurrentXPUStream().queue());
  curr_cuda_st = c10::cuda::getCurrentCUDAStream().stream();

  // CHECK: auto deviceStream = c10::xpu::getCurrentXPUStream(0);
  auto deviceStream = c10::cuda::getCurrentCUDAStream(0);

  return 0;
}
