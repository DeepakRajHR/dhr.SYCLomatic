#include "hip/hip_runtime.h"
// RUN: rm -rf %T/pytorch/ATen
// RUN: mkdir -p %T/pytorch/ATen/src
// RUN: cp %S/ATen.cu %T/pytorch/ATen/src/
// RUN: cp -r %S/pytorch_inc %T/pytorch/ATen/
// RUN: cd %T/pytorch/ATen
// RUN: mkdir dpct_out
// RUN: dpct --out-root dpct_out %T/pytorch/ATen/src/ATen.cu --extra-arg="-I%T/pytorch/ATen/pytorch_inc" --cuda-include-path="%cuda-path/include" --rule-file=%S/../../../tools/dpct/extensions/pytorch_api_rules/pytorch_api.yaml --analysis-scope-path %T/pytorch/ATen/pytorch_inc --analysis-scope-path %T/pytorch/ATen/src --in-root %T/pytorch/ATen/src
// RUN: FileCheck --input-file %T/pytorch/ATen/dpct_out/ATen.dp.cpp --match-full-lines %T/pytorch/ATen/src/ATen.cu

// CHECK: #include <c10/xpu/XPUStream.h>
#include <iostream>
// CHECK: #include <ATen/xpu/XPUContext.h>
#include <ATen/cuda/HIPContext.h>
// CHECK: #include <ATen/core/Tensor.h>
#include <ATen/core/Tensor.h>

// CHECK: #include <ATen/Tensor.h>
// CHECK-NEXT: #include <c10/util/Half.h>
#include <ATen/cuda/CUDATensorMethods.cuh>

#define AT_CUDA_CHECK(stmt)  (stmt)

// CHECK: #define BE_AT_CHECK
#define BE_AT_CHECK AT_CUDA_CHECK


__global__ void kernel() {}

void test_CUDAStream_as_arg() {
  dim3 gridSize(2, 2, 1);
  dim3 blockSize(8, 8, 1);
  void *args[] = {nullptr}; 

  // CHECK: ([&]() {
  // CHECK-NEXT:   ((sycl::queue *)(c10::xpu::getCurrentXPUStream()))
  // CHECK-NEXT:       ->parallel_for(sycl::nd_range<3>(gridSize * blockSize, blockSize),
  // CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:                        kernel();
  // CHECK-NEXT:                      });
  // CHECK-NEXT:   return 0;
  // CHECK-NEXT: }());
  AT_CUDA_CHECK(hipLaunchKernel((const void *)kernel, gridSize, blockSize, args, 0, at::cuda::getCurrentCUDAStream()));
}

int main() {
  // CHECK: dpct::queue_ptr st =
  // CHECK-NEXT: &static_cast<sycl::queue &>(c10::xpu::getCurrentXPUStream());
  hipStream_t st = 0;

  // stream APIs
  at::DeviceIndex devInd = 1;

  // CHECK: auto currentStream = c10::xpu::getCurrentXPUStream();
  auto currentStream = at::cuda::getCurrentCUDAStream();
  // CHECK: auto deviceStream = c10::xpu::getCurrentXPUStream(devInd);
  auto deviceStream = at::cuda::getCurrentCUDAStream(devInd);

  // CHECK: dpct::queue_ptr curr_cuda_st =
  // CHECK-NEXT:    &static_cast<sycl::queue &>(c10::xpu::getCurrentXPUStream().queue());
  hipStream_t curr_cuda_st = at::cuda::getCurrentCUDAStream().stream();
  // CHECK: dpct::queue_ptr dev_cuda_st = &static_cast<sycl::queue &>(
  // CHECK-NEXT:    c10::xpu::getCurrentXPUStream(devInd).queue());
  hipStream_t dev_cuda_st = at::cuda::getCurrentCUDAStream(devInd).stream();

  test_CUDAStream_as_arg();

  return 0;
}
