// RUN: rm -rf %T/pytorch/ATen
// RUN: mkdir -p %T/pytorch/ATen/src
// RUN: cp %S/ATen.cu %T/pytorch/ATen/src/
// RUN: cp %S/user_defined_rule_pytorch.yaml %T/pytorch/ATen/
// RUN: cp -r %S/pytorch_cuda_inc %T/pytorch/ATen/
// RUN: cd %T/pytorch/ATen
// RUN: mkdir dpct_out
// RUN: dpct -out-root dpct_out %T/pytorch/ATen/src/ATen.cu --extra-arg="-I%T/pytorch/ATen/pytorch_cuda_inc" --cuda-include-path="%cuda-path/include" --rule-file=%T/pytorch/ATen/user_defined_rule_pytorch.yaml  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/pytorch/ATen/dpct_out/ATen.dp.cpp --match-full-lines %T/pytorch/ATen/src/ATen.cu
// RUN: %if build_lit %{icpx -c -fsycl -DNO_BUILD_TEST  %T/pytorch/ATen/dpct_out/ATen.dp.cpp -o %T/pytorch/ATen/dpct_out/ATen.dp.o %}

#ifndef NO_BUILD_TEST
#include <iostream>
// CHECK: #include <ATen/xpu/XPUContext.h>
#include <ATen/cuda/HIPContext.h>
// CHECK: #include <ATen/core/Tensor.h>
#include <ATen/core/Tensor.h>

int main() {

  return 0;
}
#endif
