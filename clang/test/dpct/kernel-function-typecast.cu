// RUN: dpct -out-root %T/kernel-function-typecast %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/kernel-function-typecast/kernel-function-typecast.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/kernel-function-typecast/kernel-function-typecast.dp.cpp -o %T/kernel-function-typecast/kernel-function-typecast.dp.o %}

#include <cstdint>
#include <hip/hip_runtime.h>

typedef uint64_t u64;
u64 foo(hipFunction_t cuFunc, hipModule_t cuMod) {
  hipModuleGetFunction(&cuFunc, cuMod, "kfoo");
  u64 function = (u64)cuFunc;
  return function;
}
