// RUN: dpct -out-root %T/kernel-function-typecast %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/kernel-function-typecast/kernel-function-typecast.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/kernel-function-typecast/kernel-function-typecast.dp.cpp -o %T/kernel-function-typecast/kernel-function-typecast.dp.o %}

#include <cstdint>
#include <hip/hip_runtime.h>

typedef uint64_t u64;

// CHECK: void exec_kernel(dpct::kernel_function cuFunc, dpct::kernel_library cuMod, dpct::queue_ptr stream) {
void exec_kernel(hipFunction_t cuFunc, hipModule_t cuMod, hipStream_t stream) {
  u64 mod;
  u64 function;

  // verify the conversion from dpct::kernel_library to uint64_t
  mod = (u64)cuMod;

  // verify the conversion from uint64_t to dpct::kernel_library
  // CHECK: cuFunc = dpct::get_kernel_function((dpct::kernel_library)mod, "kfoo");
  hipModuleGetFunction(&cuFunc, (hipModule_t)mod, "kfoo");

  // verify the conversion from dpct::kernel_function to uint64_t
  function = (u64)cuFunc;

  void *config[] = {0};

  // verify the conversion from uint64_t to dpct::kernel_function
  // CHECK: dpct::invoke_kernel_function((dpct::kernel_function)function, *stream, sycl::range<3>(100, 100, 100), sycl::range<3>(100, 100, 100), 1024, NULL, config);
  hipModuleLaunchKernel((hipFunction_t)function, 100, 100, 100, 100, 100, 100, 1024, stream, NULL, config);
}
