
#include <hip/hip_runtime.h>
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuGraphicsMapResources | FileCheck %s -check-prefix=CU_GRAPHICS_MAP_RESOURCES
// CU_GRAPHICS_MAP_RESOURCES: CUDA API:
// CU_GRAPHICS_MAP_RESOURCES-NEXT:    cuGraphicsMapResources(c /*int*/,
// CU_GRAPHICS_MAP_RESOURCES-NEXT:                           r /*CUgraphicsResource **/,
// CU_GRAPHICS_MAP_RESOURCES-NEXT:                           s /*CUstream*/);
// CU_GRAPHICS_MAP_RESOURCES-NEXT: Is migrated to (with the option --use-experimental-features=bindless_images):
// CU_GRAPHICS_MAP_RESOURCES-NEXT:    dpct::experimental::map_resources(c, r, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuGraphicsResourceGetMappedPointer | FileCheck %s -check-prefix=CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER
// CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER: CUDA API:
// CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER-NEXT:    cuGraphicsResourceGetMappedPointer(&ptr /*CUdeviceptr **/,
// CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER-NEXT:                                       s /*size_t **/,
// CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER-NEXT:                                       r /*CUgraphicsResource*/);
// CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER-NEXT: Is migrated to (with the option --use-experimental-features=bindless_images):
// CU_GRAPHICS_RESOURCE_GET_MAPPED_POINTER-NEXT:    r->get_mapped_pointer((void **)&ptr, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuGraphicsUnmapResources | FileCheck %s -check-prefix=CU_GRAPHICS_UNMAP_RESOURCES
// CU_GRAPHICS_UNMAP_RESOURCES: CUDA API:
// CU_GRAPHICS_UNMAP_RESOURCES-NEXT:    cuGraphicsUnmapResources(c /*int*/,
// CU_GRAPHICS_UNMAP_RESOURCES-NEXT:                             r /*CUgraphicsResource **/,
// CU_GRAPHICS_UNMAP_RESOURCES-NEXT:                             s /*CUstream*/);
// CU_GRAPHICS_UNMAP_RESOURCES-NEXT: Is migrated to (with the option --use-experimental-features=bindless_images):
// CU_GRAPHICS_UNMAP_RESOURCES-NEXT:    dpct::experimental::unmap_resources(c, r, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuGraphicsUnregisterResource | FileCheck %s -check-prefix=CU_GRAPHICS_UNREGISTER_RESOURCE
// CU_GRAPHICS_UNREGISTER_RESOURCE: CUDA API:
// CU_GRAPHICS_UNREGISTER_RESOURCE-NEXT:    cuGraphicsUnregisterResource(r /*CUgraphicsResource*/);
// CU_GRAPHICS_UNREGISTER_RESOURCE-NEXT: Is migrated to (with the option --use-experimental-features=bindless_images):
// CU_GRAPHICS_UNREGISTER_RESOURCE-NEXT:    delete r;
