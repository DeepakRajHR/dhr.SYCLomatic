
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T/miscellaneous_apis %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/miscellaneous_apis/miscellaneous_apis.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/miscellaneous_apis/miscellaneous_apis.dp.cpp -o %T/miscellaneous_apis/miscellaneous_apis.dp.o %}

void foo(const void **table, hipUUID *tableId) {
  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuGetExportTable was removed because the functionality is unknown for the undocumented API.
  // CHECK-NEXT: */
  cuGetExportTable(table, tableId);
}
