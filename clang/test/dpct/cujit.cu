
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T/USM-restricted %s --cuda-include-path="%cuda-path/include" -- -x cuda
// RUN: FileCheck --match-full-lines --input-file %T/cujit.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cujit.dp.cpp -o %T/cujit.dp.o %}

void foo(CUjit_target &target, hipJitInputType &type, int vMajor, int vMinor) {
  if (vMajor == 5 && vMinor < 2) {
    target = CU_TARGET_COMPUTE_50;
    type = hipJitInputCubin;
  } else {
    target = CU_TARGET_COMPUTE_52;
    type = hipJitInputPtx;
  }
}
