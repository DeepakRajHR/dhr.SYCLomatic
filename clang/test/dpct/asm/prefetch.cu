// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/prefetch %s --use-experimental-features=prefetch --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/prefetch/prefetch.dp.cpp
// RUN: %if BUILD_LIT %{icpx -c -DBUILD_TEST -fsycl %T/prefetch/prefetch.dp.cpp -o %T/prefetch/prefetch.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>

/*
Supported syntax:
-----------------
prefetch.level [a];                            // prefetch to generic addr space cache
prefetch.global.level [a];                     // prefetch to global cache

Unsupported syntax:
-------------------
prefetch.local.level
prefetch.global.level::eviction_priority [a];   // prefetch to data cache
prefetch{.tensormap_space}.tensormap [a];       // prefetch the tensormap

.level =                    { .L1, .L2 };
.level::eviction_priority = { .L2::evict_last, .L2::evict_normal };
.tensormap_space =          { .const, .param };
*/

__global__ void prefetch(int *arr) {
  /* prefetch of no address space */
  // CHECK: sycl::ext::oneapi::experimental::prefetch(arr, sycl::ext::oneapi::experimental::properties{sycl::ext::oneapi::experimental::prefetch_hint_L1});
  asm volatile ("prefetch.L1 [%0];" : : "l"(arr));
  // CHECK: sycl::ext::oneapi::experimental::prefetch(arr, sycl::ext::oneapi::experimental::properties{sycl::ext::oneapi::experimental::prefetch_hint_L2});
  asm volatile ("prefetch.L2 [%0];" : : "l"(arr));

  /* prefetch of global address space */
  // CHECK: sycl::ext::oneapi::experimental::prefetch(arr, sycl::ext::oneapi::experimental::properties{sycl::ext::oneapi::experimental::prefetch_hint_L1});
  asm volatile ("prefetch.global.L1 [%0];" : : "l"(arr));
  // CHECK: sycl::ext::oneapi::experimental::prefetch(arr, sycl::ext::oneapi::experimental::properties{sycl::ext::oneapi::experimental::prefetch_hint_L2});
  asm volatile ("prefetch.global.L2 [%0];" : : "l"(arr));

  /* using Register-Immediate (Displacement) address mode */
  // CHECK: sycl::ext::oneapi::experimental::prefetch(((uint8_t *)((uintptr_t)arr + 2)), sycl::ext::oneapi::experimental::properties{sycl::ext::oneapi::experimental::prefetch_hint_L1});
  asm volatile("prefetch.global.L1 [%0 + 2];" :: "l"(arr));
}

// clang-format on
