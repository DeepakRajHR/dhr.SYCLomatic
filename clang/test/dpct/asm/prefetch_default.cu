// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/prefetch_default %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/prefetch_default/prefetch_default.dp.cpp
// RUN: %if BUILD_LIT %{icpx -c -DBUILD_TEST -fsycl %T/prefetch_default/prefetch_default.dp.cpp -o %T/prefetch_default/prefetch_default.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>

__global__ void prefetch(int *arr) {
#ifndef BUILD_TEST
  /* prefetch of no address space */
  // CHECK: /*
  // CHECK-NEXT: DPCT1053:{{.*}} Migration of device assembly code is not supported.
  // CHECK-NEXT: */
  asm volatile ("prefetch.L1 [%0];" : : "l"(arr));
  // CHECK: /*
  // CHECK-NEXT: DPCT1053:{{.*}} Migration of device assembly code is not supported.
  // CHECK-NEXT: */
  asm volatile ("prefetch.L2 [%0];" : : "l"(arr));

  /* prefetch of global address space */
  // CHECK: /*
  // CHECK-NEXT: DPCT1053:{{.*}} Migration of device assembly code is not supported.
  // CHECK-NEXT: */
  asm volatile ("prefetch.global.L1 [%0];" : : "l"(arr));
  // CHECK: /*
  // CHECK-NEXT: DPCT1053:{{.*}} Migration of device assembly code is not supported.
  // CHECK-NEXT: */
  asm volatile ("prefetch.global.L2 [%0];" : : "l"(arr));

  /* using Register-Immediate (Displacement) address mode */
  // CHECK: /*
  // CHECK-NEXT: DPCT1053:{{.*}} Migration of device assembly code is not supported.
  // CHECK-NEXT: */
  asm volatile("prefetch.global.L1 [%0 + 2];" :: "l"(arr));
#endif // BUILD_TEST
}

// clang-format on
