// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --use-experimental-features=bindless_images --format-range=none -out-root %T/cuGraphics %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuGraphics/cuGraphics.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -DNO_BUILD_TEST -fsycl %T/cuGraphics/cuGraphics.dp.cpp -o %T/cuGraphics/cuGraphics.dp.o %}

#include <hip/hip_runtime.h>

int main() {
  // CHECK: dpct::experimental::external_mem_wrapper_ptr resource;
  // CHECK-NEXT: dpct::experimental::external_mem_wrapper_ptr *resources;
  // CHECK-NEXT: dpct::experimental::external_mem_wrapper_ptr **resources_ptr;
  hipGraphicsResource_t resource;
  hipGraphicsResource_t *resources;
  hipGraphicsResource_t **resources_ptr;

  // CHECK: dpct::experimental::external_mem_wrapper_ptr resources_arr[10];
  hipGraphicsResource_t resources_arr[10];

  // CHECK: dpct::experimental::external_mem_wrapper_ptr resource1, *resources1, **resources_ptr1;
  hipGraphicsResource_t resource1, *resources1, **resources_ptr1;

  resources_arr[0] = resource;
  resources_arr[1] = resource1;

  hipDeviceptr_t pDevPtr;
  size_t pSize;

  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, 0);

#ifdef _WIN32
  // CHECK-WINDOWS: dpct::experimental::map_resources(2, resources_arr, stream);
  cuGraphicsMapResources(2, resources_arr, stream);

  // CHECK-WINDOWS: dpct::experimental::map_resources(1, &resource, stream);
  cuGraphicsMapResources(1, &resource, stream);
#endif // _WIN32

  // CHECK: resource->get_mapped_pointer((void **)&pDevPtr, &pSize);
  hipGraphicsResourceGetMappedPointer(&pDevPtr, &pSize, resource);

#ifdef _WIN32
  // CHECK-WINDOWS: dpct::experimental::unmap_resources(2, resources_arr, stream);
  cuGraphicsUnmapResources(2, resources_arr, stream);

  // CHECK-WINDOWS: dpct::experimental::unmap_resources(1, &resource, stream);
  cuGraphicsUnmapResources(1, &resource, stream);
#endif // _WIN32

  // CHECK: delete resource;
  hipGraphicsUnregisterResource(resource);

  // CHECK: delete resource1;
  hipGraphicsUnregisterResource(resource1);

  return 0;
}
