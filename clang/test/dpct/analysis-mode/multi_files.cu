// RUN: dpct %S/multi_files.cu -in-root=%S --cuda-include-path="%cuda-path/include" -analysis-mode -analysis-mode-output-file=%T/multi_report.out

// RUN: echo "// CHECK-DAG: {{.*}}multi_files.cu:" > %T/multi_files.check
// RUN: echo "// CHECK-DAG: {{.*}}multi_files.h:" >> %T/multi_files.check
// RUN: cat %S/multi_files.check >> %T/multi_files.check

// RUN: FileCheck --match-full-lines --input-file %T/multi_report.out %T/multi_files.check

#include "multi_files.h"

void foo() {
  int *a;
  hipDeviceGetPCIBusId(nullptr, 0, 0);
  hipMalloc(&a, sizeof(int) * 4);
}