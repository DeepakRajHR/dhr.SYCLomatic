// RUN: dpct %S/multi_files.cu -in-root=%S -out-root=%T/multi -analysis-mode -analysis-mode-output-file=%T/multi_report.out

// RUN: echo "// CHECK: %S/multi_files.cu:" > %T/multi_files.check
// RUN: echo "// CHECK: %S/multi_files.h:" >> %T/multi_files.check
// RUN: cat %S/multi_files.check >> %T/multi_files.check

// RUN: FileCheck --match-full-lines --input-file %T/multi_report.out %T/multi_files.check

#include "multi_files.h"

void foo() {
  int *a;
  hipDeviceGetPCIBusId(nullptr, 0, 0);
  hipMalloc(&a, sizeof(int) * 4);
}