// RUN: dpct -in-root=%S -analysis-mode --cuda-include-path="%cuda-path/include" -analysis-mode-output-file=%T/single_file.out %S/single_file.cu

// RUN: echo "// CHECK: {{.*}}single_file.cu:" > %T/single_file.check
// RUN: cat %S/single_file.check >> %T/single_file.check

// RUN: FileCheck --match-full-lines --input-file %T/single_file.out %T/single_file.check 


#include <hip/hip_runtime.h>
#include "cudnn.h"

void foo() {
  int *a;
  cudnnReduceTensorIndices_t m;
  hipDeviceGetPCIBusId(nullptr, 0, 0);
  hipDeviceGetPCIBusId(nullptr, 0, 0);
  hipMalloc(&a, sizeof(int) * 4);
}