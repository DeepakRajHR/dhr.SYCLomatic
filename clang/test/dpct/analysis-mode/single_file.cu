// RUN: dpct -in-root=%S -out-root=%T/single -analysis-mode -analysis-mode-output-file=%T/single_file.out %S/single_file.cu

// RUN: echo "// CHECK: %S/single_file.cu:" > %T/single_file.check
// RUN: cat %S/single_file.check >> %T/single.check

// RUN: FileCheck --match-full-lines --input-file %T/single_file.out %T/single_file.check 


#include <hip/hip_runtime.h>
#include "cudnn.h"

void foo() {
  int *a;
  cudnnReduceTensorIndices_t m;
  hipDeviceGetPCIBusId(nullptr, 0, 0);
  hipDeviceGetPCIBusId(nullptr, 0, 0);
  hipMalloc(&a, sizeof(int) * 4);
}